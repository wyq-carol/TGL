
#include "../util/computeUtil.h"
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <torch/types.h>

void csr2cscKernel(int m, int n, int nnz, int *csrRowPtr, int *csrColInd,
                   float *csrVal, int *cscColPtr, int *cscRowInd,
                   float *cscVal) {
  hipsparseHandle_t handle;
  checkCuSparseError(hipsparseCreate(&handle));
  size_t bufferSize = 0;
  void *buffer = NULL;
  checkCuSparseError(hipsparseCsr2cscEx2_bufferSize(
      handle, m, n, nnz, csrVal, csrRowPtr, csrColInd, cscVal, cscColPtr,
      cscRowInd, HIP_R_32F, HIPSPARSE_ACTION_SYMBOLIC, HIPSPARSE_INDEX_BASE_ZERO,
      HIPSPARSE_CSR2CSC_ALG1, &bufferSize));
  checkCudaError(hipMalloc((void **)&buffer, bufferSize * sizeof(float)));
  checkCuSparseError(hipsparseCsr2cscEx2(
      handle, m, n, nnz, csrVal, csrRowPtr, csrColInd, cscVal, cscColPtr,
      cscRowInd, HIP_R_32F, HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO,
      HIPSPARSE_CSR2CSC_ALG1, buffer));
  checkCudaError(hipFree(buffer));
}

std::vector<torch::Tensor> csr2csc_cuda(torch::Tensor csrRowPtr,
                                        torch::Tensor csrColInd,
                                        torch::Tensor csrVal) {
  const auto n = csrRowPtr.size(0) - 1;
  const auto nnz = csrColInd.size(0);
  auto devid = csrRowPtr.device().index();
  auto optionsF =
      torch::TensorOptions().dtype(torch::kFloat32).device(torch::kCUDA, devid);
  auto optionsI =
      torch::TensorOptions().dtype(torch::kInt32).device(torch::kCUDA, devid);
  auto cscColPtr = torch::empty({n + 1}, optionsI);
  auto cscRowInd = torch::empty({nnz}, optionsI);
  auto cscVal = torch::empty({nnz}, optionsF);
  csr2cscKernel(n, n, nnz, csrRowPtr.data_ptr<int>(), csrColInd.data_ptr<int>(),
                csrVal.data_ptr<float>(), cscColPtr.data_ptr<int>(),
                cscRowInd.data_ptr<int>(), cscVal.data_ptr<float>());
  return {cscColPtr, cscRowInd, cscVal};
}